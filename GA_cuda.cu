#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define STR_LENGTH 60
#define CHILD_NUM_PER_GENERATION 20000


/*please note that this program treat type char as signed, different platform may produce different result */
typedef struct offspring{
	char str[STR_LENGTH];
	int score;
}offspring;
offspring pa[CHILD_NUM_PER_GENERATION];
offspring ch[CHILD_NUM_PER_GENERATION];
offspring *tmp,*out;

const int mutation_rate = 100;/*produces mutation with a probability of 1/100*/

void string_random_create(offspring* input){
	int i;
	for(i=0;i<STR_LENGTH;i++){
		input->str[i] = (rand()%95 + 32);/*choose character from 32 to 126*/
	}
	return;
}

__host__ __device__ void fitness(offspring* input){/*this is the fitness function that will score the child */
	int i;
	char target[STR_LENGTH+1] = "Hello World!Hello World!Hello World!Hello World!Hello World!";
	input->score = 0;
	for(i=0;i<STR_LENGTH;i++){
		input->score -= abs(input->str[i] - target[i]);
	}

	return;
}


__device__ void mutation(offspring* input,int data){
	int i;
	for(i=0;i<STR_LENGTH;i++){
		if(data%mutation_rate == 0){
			/*mutate*/
			input->str[i] = data%95 + 32;
		}
		/*don't mutate*/
	}
}


int print_best_result(int generation, offspring* child){
	int result = -1;
	int i;
	int score = -1000000;/*setting to -1000000 as it is an impossible score to reach */
	for(i=0;i<CHILD_NUM_PER_GENERATION;i++){
		if(child[i].score > score){
			score = child[i].score;
			result = i;
		}
	}
	printf("The best score in %d turn is %d : str= ", generation, score);
	for(i=0;i<STR_LENGTH;i++){
		printf("%c",child[result].str[i]);
	}
	printf("\n");
	return score;
}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
}


__device__ char * my_strcpy(char *dest, const char *src){
  int i = 0;
  do {
    dest[i] = src[i];}
  while (src[i++] != 0);
  return dest;
}

__global__ void kernel4(hiprandState* State,offspring *famo,offspring *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
	int j;
	int parent1 = -1;
	int parent2 = -1;
	int parent3 = -1;
	int parent4 = -1;
	int better1;
	int better2;
	
    parent1=fabsf(hiprand(&State[idx*4])%CHILD_NUM_PER_GENERATION);
    parent2=fabsf(hiprand(&State[idx*4+1])%CHILD_NUM_PER_GENERATION);
    parent3=fabsf(hiprand(&State[idx*4+2])%CHILD_NUM_PER_GENERATION);
    parent4=fabsf(hiprand(&State[idx*4+3])%CHILD_NUM_PER_GENERATION);

    if(famo[parent1].score > famo[parent2].score){
		better1 = parent1;
	}
	else{
		better1 = parent2;
	}
	if(famo[parent3].score > famo[parent4].score){
		better2 = parent3;
	}
	else{
		better2 = parent4;
	}
	
    i = fabsf(hiprand(&State[idx*2])%(STR_LENGTH+1));
    for(j=0;j<i;j++){
		output[idx*4].str[j] = famo[better1].str[j];
		output[idx*4+1].str[j] = famo[better2].str[j];
	}
	for(j=i;j<STR_LENGTH;j++){
		output[idx*4].str[j] = famo[better2].str[j];
		output[idx*4+1].str[j] = famo[better1].str[j];
	}
	i = fabsf(hiprand(&State[idx*2+1])%(STR_LENGTH+1));
	for(j=0;j<i;j++){
		output[idx*4+2].str[j] = famo[better1].str[j];
		output[idx*4+3].str[j] = famo[better2].str[j];
	}
	for(j=i;j<STR_LENGTH;j++){
		output[idx*4+2].str[j] = famo[better2].str[j];
		output[idx*4+3].str[j] = famo[better1].str[j];
	}
    mutation(&output[idx*4],parent1);
    mutation(&output[idx*4+1],parent2);
    mutation(&output[idx*4+2],parent3);
    mutation(&output[idx*4+3],parent4);
    fitness(&output[idx*4]);
    fitness(&output[idx*4+1]);
    fitness(&output[idx*4+2]);
    fitness(&output[idx*4+3]);
}

int main(){
	srand(time(NULL));
	int i;
	printf("See how the string evolves into \"Hello World!Hello World!Hello World!Hello World!Hello World!\"\n");

	memset(&pa, 0, CHILD_NUM_PER_GENERATION*sizeof(offspring));
	memset(&ch, 0, CHILD_NUM_PER_GENERATION*sizeof(offspring));
	for(i=0;i<CHILD_NUM_PER_GENERATION;i++){
		string_random_create(&pa[i]);/*these will be the first parents*/
		fitness(&pa[i]);
	}
	print_best_result(0,pa);

	////cuda
    hiprandState* devStates;
    hipMalloc ( &devStates, 25000*sizeof( hiprandState ) );
	hipMalloc((void**)&tmp, CHILD_NUM_PER_GENERATION*sizeof(offspring));
	hipMalloc((void**)&out, CHILD_NUM_PER_GENERATION*sizeof(offspring));

    hipMemcpy(tmp, pa, CHILD_NUM_PER_GENERATION*sizeof(offspring), hipMemcpyHostToDevice);
	for(i=1;i<10001;i++){
		setup_kernel <<< 40, 500 >>> (devStates,clock());
		kernel4 << <10, 500 >> >(devStates,tmp,out);
        hipDeviceSynchronize();
		hipMemcpy(tmp, out, CHILD_NUM_PER_GENERATION*sizeof(offspring), hipMemcpyDeviceToDevice);
		//hipMemcpy(ch, out, CHILD_NUM_PER_GENERATION*sizeof(offspring), hipMemcpyDeviceToHost);
        //print_best_result(i,ch);
	}
	hipMemcpy(ch, out, CHILD_NUM_PER_GENERATION*sizeof(offspring), hipMemcpyDeviceToHost);
	hipFree(&tmp);
	hipFree(&out);
	////////////////////////////
	print_best_result(10000,ch);
	return 0;
}
